#include "hip/hip_runtime.h"
#include "nvmatrix/include/nvmatrix_kernels.cuh"
#include "nvmatrix/include/nvmatrix_operators.cuh"
#include <THC.h>
#include <algorithm>
#include "hip/hip_runtime_api.h"

#define THAssert(exp)  if (exp) ; \
  else THError("assert(%s) failed in file %s, line %d", #exp, __FILE__, __LINE__);

extern "C" {
  
  void addBias(THCudaTensor* output, THCudaTensor* bias) {
    int width = output->size[1];
    int height = output->size[0];
    float *odata = THCudaTensor_data(output);
    float *bdata = THCudaTensor_data(bias);
    dim3 threads(ADD_VEC_THREADS_X, ADD_VEC_THREADS_Y);
    dim3 blocks(std::min(512, DIVUP(width, ADD_VEC_THREADS_X)),
                std::min(NUM_BLOCKS_MAX, DIVUP(height, ADD_VEC_THREADS_Y)));
    kColVectorOp<NVMatrixBinaryOps::Add>
      <<<blocks, threads>>>(odata, bdata, odata, width, height, 
                            output->stride[0], output->stride[0], 
                            NVMatrixBinaryOps::Add());
    getLastCudaError("Kernel execution failed");
  }
  
  void gradBias(THCudaTensor* gradOutput, THCudaTensor* gradBias, float scale) {
    dim3 threads(AWR_NUM_THREADS);
    dim3 blocks(1, gradOutput->size[0]);
    kAggRows_wholerow_nosync<<<blocks, threads>>>(THCudaTensor_data(gradOutput), THCudaTensor_data(gradBias), gradOutput->size[1], gradOutput->size[0], NVMatrixAggs::Sum(), NVMatrixOps::Identity(), NVMatrixBinaryOps::SecondScaled(scale));
  }

  void addSumCols(THCudaTensor*output, THCudaTensor*input) { // output = weights, input = wtemp
    int width = input->size[1];
    int height = input->size[0];
    THCudaTensor_resize2d(output, 1, width);
    if ((height <= 2048 || width >= 4096)) {
      int numBlocks = DIVUP(width, NUM_SUM_COLS_THREADS_PER_BLOCK);
      THAssert(numBlocks * NUM_SUM_COLS_THREADS_PER_BLOCK >= width);
      THAssert(numBlocks < NUM_BLOCKS_MAX);
      hipTextureObject_t texInput = THCudaTensor_getTextureObject(input);
      kDumbAggCols<NVMatrixAggs::Sum, NVMatrixOps::Identity, NVMatrixBinaryOps::SecondScaled>
        <<<numBlocks,NUM_SUM_COLS_THREADS_PER_BLOCK, 0>>>(texInput, 
                                                          THCudaTensor_data(output), width, height,
                                                          NVMatrixAggs::Sum(), NVMatrixOps::Identity(), 
                                                          NVMatrixBinaryOps::SecondScaled(1.0));
      getLastCudaError("kDumbAggCols: Kernel execution failed");
      checkCudaErrors(hipDestroyTextureObject(texInput));
    } else { // Specialize the case when we have very long columns and few of them
      const int sumLength = 128;
      THCudaTensor* tmp = THCudaTensor_newWithSize2d(DIVUP(height, sumLength), width);
      int numBlocksX = DIVUP(width, NUM_SUM_COLS_THREADS_PER_BLOCK);
      int numBlocksY = DIVUP(height, sumLength);
      dim3 blocks(numBlocksX, numBlocksY);
      dim3 threads(NUM_SUM_COLS_THREADS_PER_BLOCK);
      hipTextureObject_t texInput = THCudaTensor_getTextureObject(input);
      kAggCols<NVMatrixAggs::Sum, NVMatrixOps::Identity>
        <<<blocks,threads, 0>>>(texInput, THCudaTensor_data(tmp), 
                                width, height, sumLength, NVMatrixAggs::Sum(), NVMatrixOps::Identity());
      getLastCudaError("kAggCols: Kernel execution failed");
      checkCudaErrors(hipDestroyTextureObject(texInput));

      int numBlocks = DIVUP(width, NUM_SUM_COLS_THREADS_PER_BLOCK);
      hipTextureObject_t texTmp = THCudaTensor_getTextureObject(tmp);
      kDumbAggCols<NVMatrixAggs::Sum, NVMatrixOps::Identity, NVMatrixBinaryOps::SecondScaled>
        <<<numBlocks,NUM_SUM_COLS_THREADS_PER_BLOCK, 0>>>(texTmp, THCudaTensor_data(output), width, height,
                                                          NVMatrixAggs::Sum(), NVMatrixOps::Identity(), 
                                                          NVMatrixBinaryOps::SecondScaled(1.0));
      getLastCudaError("kDumbAggCols: Kernel execution failed");
      checkCudaErrors(hipDestroyTextureObject(texTmp));
      THCudaTensor_free(tmp);
    }
  }
}
